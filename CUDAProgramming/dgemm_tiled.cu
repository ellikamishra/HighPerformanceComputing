#include "hip/hip_runtime.h"

#include "common.cuh"

template<int TILE>
__global__ void dgemm_tiled_kernel(const double* __restrict__ A,
                                   const double* __restrict__ B,
                                   double* __restrict__ C,
                                   int M, int N, int K)
{
  __shared__ double As[TILE][TILE];
  __shared__ double Bs[TILE][TILE];

  int row = blockIdx.y * TILE + threadIdx.y;
  int col = blockIdx.x * TILE + threadIdx.x;

  double acc = 0.0;

  // Number of tiles over K dimension
  int numTiles = (K + TILE - 1) / TILE;
  for (int t=0; t<numTiles; ++t) {
    int Acol = t*TILE + threadIdx.x;
    int Brow = t*TILE + threadIdx.y;

    // Load tiles with bounds checks
    As[threadIdx.y][threadIdx.x] = (row < M && Acol < K)
        ? A[(size_t)row*K + Acol] : 0.0;
    Bs[threadIdx.y][threadIdx.x] = (Brow < K && col < N)
        ? B[(size_t)Brow*N + col] : 0.0;

    __syncthreads();

    for (int k=0; k<TILE; ++k) {
      acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }
    __syncthreads();
  }

  if (row < M && col < N) {
    C[(size_t)row*N + col] = acc;
  }
}

// Host launcher selecting TILE at runtime
void dgemm_tiled_gpu(const double* dA, const double* dB, double* dC,
                     int M, int N, int K, int tile, hipStream_t stream)
{
  // choose from supported tile sizes
  dim3 block, grid;
  auto launch = [&](auto kernel){
    block = dim3(tile, tile);
    grid = dim3( (N + tile - 1)/tile, (M + tile - 1)/tile );
    kernel<<<grid, block, 0, stream>>>(dA, dB, dC, M, N, K);
    CUDA_CHECK(hipGetLastError());
  };

  switch (tile) {
    case 1:  launch(dgemm_tiled_kernel<1>);  break;
    case 4:  launch(dgemm_tiled_kernel<4>);  break;
    case 8:  launch(dgemm_tiled_kernel<8>);  break;
    case 16: launch(dgemm_tiled_kernel<16>); break;
    case 32: launch(dgemm_tiled_kernel<32>); break;
    default: // fallback to 16
      fprintf(stderr, "[WARN] Unsupported TILE=%d, using 16\n", tile);
      launch(dgemm_tiled_kernel<16>);
  }
}
