#include "hip/hip_runtime.h"

#include <cstdio>
#include <vector>
#include <random>
#include <cstring>
#include "common.cuh"

void dgemm_naive_gpu (const double*, const double*, double*, int,int,int, hipStream_t);
void dgemm_tiled_gpu (const double*, const double*, double*, int,int,int, int, hipStream_t);
void attention_via_dgemm(const double*, const double*, const double*, double*, int,int, int, hipblasHandle_t, hipStream_t);

static int argi(char** a, int &i){ return std::atoi(a[++i]); }

int main(int argc, char** argv) {
  // Defaults smaller; override in SLURM scripts
  int M=2048, N=2048, K=2048;
  int L=2048, D=512;
  int task=0; // 0:naive, 1:tiled, 2:cublas, 3:attn-naive, 4:attn-tiled, 5:attn-cublas
  int TILE=32;

  for (int i=1;i<argc;i++) {
    if (!strcmp(argv[i],"--task") && i+1<argc) task = argi(argv, i);
    else if (!strcmp(argv[i],"--M") && i+1<argc) M = argi(argv, i);
    else if (!strcmp(argv[i],"--N") && i+1<argc) N = argi(argv, i);
    else if (!strcmp(argv[i],"--K") && i+1<argc) K = argi(argv, i);
    else if (!strcmp(argv[i],"--L") && i+1<argc) L = argi(argv, i);
    else if (!strcmp(argv[i],"--D") && i+1<argc) D = argi(argv, i);
    else if (!strcmp(argv[i],"--TILE") && i+1<argc) TILE = argi(argv, i);
  }

  CUDA_CHECK(hipSetDevice(0));
  hipblasHandle_t h; CUBLAS_CHECK(hipblasCreate(&h));
  hipStream_t stream; CUDA_CHECK(hipStreamCreate(&stream));
  CUBLAS_CHECK(hipblasSetStream(h, stream));

  std::vector<double> hA((size_t)M*K), hB((size_t)K*N), hC((size_t)M*N, 0.0), hCref((size_t)M*N,0.0);
  std::vector<double> hQ((size_t)L*D), hK((size_t)L*D), hV((size_t)L*D), hO((size_t)L*D), hOref((size_t)L*D);

  fill_host(hA.data(), hA.size(), 1); fill_host(hB.data(), hB.size(), 2);
  fill_host(hQ.data(), hQ.size(), 3); fill_host(hK.data(), hK.size(), 4); fill_host(hV.data(), hV.size(), 5);

  double *dA=nullptr, *dB=nullptr, *dC=nullptr;
  double *dQ=nullptr, *dK=nullptr, *dV=nullptr, *dO=nullptr, *dCref=nullptr, *dOref=nullptr;
  CUDA_CHECK(hipMalloc(&dA, hA.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dB, hB.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dC, hC.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dCref, hCref.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dQ, hQ.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dK, hK.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dV, hV.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dO, hO.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dOref, hOref.size()*sizeof(double)));

  CUDA_CHECK(hipMemcpyAsync(dA, hA.data(), hA.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dB, hB.data(), hB.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dQ, hQ.data(), hQ.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dK, hK.data(), hK.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dV, hV.data(), hV.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemsetAsync(dC, 0, hC.size()*sizeof(double), stream));
  CUDA_CHECK(hipMemsetAsync(dO, 0, hO.size()*sizeof(double), stream));
  CUDA_CHECK(hipStreamSynchronize(stream));

  // cuBLAS references
  cublas_rowmajor_dgemm(h, M, N, K, dA, dB, dCref);
  attention_via_dgemm(dQ, dK, dV, dOref, L, D, 2, h, stream);

  hipEvent_t e0, e1; CUDA_CHECK(hipEventCreate(&e0)); CUDA_CHECK(hipEventCreate(&e1));

  if (task == 0) {
    CUDA_CHECK(hipEventRecord(e0, stream));
    dgemm_naive_gpu(dA, dB, dC, M, N, K, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    int bx=16, by=16; if (const char* s=getenv("NBX")) bx=std::max(1,atoi(s)); if (const char* s2=getenv("NBY")) by=std::max(1,atoi(s2));
printf("CSV,ALG=DGEMM_NAIVE,M=%d,N=%d,K=%d,TILE=-,BLOCK=%dx%d,ms=%.3f,GF=%.3f,diff=%.3e\n", M,N,K,bx,by,ms,gflops,mad);
  }
  else if (task == 1) { // tiled
    CUDA_CHECK(hipEventRecord(e0, stream));
    dgemm_tiled_gpu(dA, dB, dC, M, N, K, TILE, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    printf("CSV,ALG=DGEMM_TILED,M=%d,N=%d,K=%d,TILE=%d,BLOCK=%dx%d,ms=%.3f,GF=%.3f,diff=%.3e\n", M,N,K,TILE,TILE,TILE,ms,gflops,mad);
  }
  else if (task == 2) { // cuBLAS
    CUDA_CHECK(hipEventRecord(e0, stream));
    cublas_rowmajor_dgemm(h, M, N, K, dA, dB, dC);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    printf("CSV,ALG=DGEMM_CUBLAS,M=%d,N=%d,K=%d,TILE=-,BLOCK=-,ms=%.3f,GF=%.3f,diff=%.3e\n", M,N,K,ms,gflops,mad);
  }
  else if (task >= 3 && task <= 5) { // attention via DGEMM
    int mode = (task==3)?0: (task==4)?1:2;
    CUDA_CHECK(hipEventRecord(e0, stream));
    attention_via_dgemm(dQ, dK, dV, dO, L, D, mode, h, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hO.data(), dO, hO.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hOref.data(), dOref, hOref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hO.data(), hOref.data(), hO.size());
    double flops = 4.0 * (double)L*L*D; // approx ops in attention (2 for QK^T + 2 for A*V)
    double gflops = flops / (ms*1e6);
    const char* tag = (mode==0)?"NAIVE":(mode==1)?"TILED":"CUBLAS";
    printf("CSV,ALG=ATTN_%s,L=%d,D=%d,TILE=%d,ms=%.3f,GF=%.3f,diff=%.3e\n", tag,L,D,TILE,ms,gflops,mad);
  }

  CUDA_CHECK(hipEventDestroy(e0)); CUDA_CHECK(hipEventDestroy(e1));
  CUDA_CHECK(hipFree(dA)); CUDA_CHECK(hipFree(dB)); CUDA_CHECK(hipFree(dC));
  CUDA_CHECK(hipFree(dCref));
  CUDA_CHECK(hipFree(dQ)); CUDA_CHECK(hipFree(dK)); CUDA_CHECK(hipFree(dV));
  CUDA_CHECK(hipFree(dO)); CUDA_CHECK(hipFree(dOref));
  CUBLAS_CHECK(hipblasDestroy(h)); CUDA_CHECK(hipStreamDestroy(stream));
  return 0;
}
