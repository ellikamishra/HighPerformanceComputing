#include <cstdio>
#include <vector>
#include <random>
#include <cstring>
#include "common.cuh"

void dgemm_naive_gpu (const double*, const double*, double*, int,int,int, hipStream_t);
void dgemm_tiled_gpu (const double*, const double*, double*, int,int,int, int, hipStream_t);
void attention_via_dgemm(const double*, const double*, const double*, double*, int,int, int, hipblasHandle_t, hipStream_t);

static int argi(char** a, int &i){ return std::atoi(a[++i]); }

int main(int argc, char** argv) {
  int M=1024, N=1024, K=1024;
  int L=512, D=256;
  int task=0;
  int TILE=32;

  for (int i=1;i<argc;i++) {
    if (!strcmp(argv[i],"--task") && i+1<argc) task = argi(argv, i);
    else if (!strcmp(argv[i],"--M") && i+1<argc) M = argi(argv, i);
    else if (!strcmp(argv[i],"--N") && i+1<argc) N = argi(argv, i);
    else if (!strcmp(argv[i],"--K") && i+1<argc) K = argi(argv, i);
    else if (!strcmp(argv[i],"--L") && i+1<argc) L = argi(argv, i);
    else if (!strcmp(argv[i],"--D") && i+1<argc) D = argi(argv, i);
    else if (!strcmp(argv[i],"--TILE") && i+1<argc) TILE = argi(argv, i);
  }

  CUDA_CHECK(hipSetDevice(0));
  hipblasHandle_t h; CUBLAS_CHECK(hipblasCreate(&h));
  hipStream_t stream; CUDA_CHECK(hipStreamCreate(&stream));
  CUBLAS_CHECK(hipblasSetStream(h, stream));

  std::vector<double> hA((size_t)M*K), hB((size_t)K*N), hC((size_t)M*N, 0.0), hCref((size_t)M*N,0.0);
  std::vector<double> hQ((size_t)L*D), hK((size_t)L*D), hV((size_t)L*D), hO((size_t)L*D), hOref((size_t)L*D);

  fill_host(hA.data(), hA.size(), 1); fill_host(hB.data(), hB.size(), 2);
  fill_host(hQ.data(), hQ.size(), 3); fill_host(hK.data(), hK.size(), 4); fill_host(hV.data(), hV.size(), 5);

  double *dA=nullptr, *dB=nullptr, *dC=nullptr;
  double *dQ=nullptr, *dK=nullptr, *dV=nullptr, *dO=nullptr, *dCref=nullptr, *dOref=nullptr;
  CUDA_CHECK(hipMalloc(&dA, hA.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dB, hB.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dC, hC.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dCref, hCref.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dQ, hQ.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dK, hK.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dV, hV.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dO, hO.size()*sizeof(double)));
  CUDA_CHECK(hipMalloc(&dOref, hOref.size()*sizeof(double)));

  CUDA_CHECK(hipMemcpyAsync(dA, hA.data(), hA.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dB, hB.data(), hB.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dQ, hQ.data(), hQ.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dK, hK.data(), hK.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemcpyAsync(dV, hV.data(), hV.size()*sizeof(double), hipMemcpyHostToDevice, stream));
  CUDA_CHECK(hipMemsetAsync(dC, 0, hC.size()*sizeof(double), stream));
  CUDA_CHECK(hipMemsetAsync(dO, 0, hO.size()*sizeof(double), stream));
  CUDA_CHECK(hipStreamSynchronize(stream));


 
  cublas_rowmajor_dgemm(h, M, N, K, dA, dB, dCref);
  attention_via_dgemm(dQ, dK, dV, dOref, L, D, 2, h, stream);

  hipEvent_t e0, e1; CUDA_CHECK(hipEventCreate(&e0)); CUDA_CHECK(hipEventCreate(&e1));

  // Naive 
  if (task == 0) {
    CUDA_CHECK(hipEventRecord(e0, stream));
    dgemm_naive_gpu(dA, dB, dC, M, N, K, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    printf("[DGEMM naive] time=%.3f ms rate=%.2f GF/s diff=%.3e\n", ms,gflops,mad);
  }
  else if (task == 1) { // blocked
    CUDA_CHECK(hipEventRecord(e0, stream));
    dgemm_tiled_gpu(dA, dB, dC, M, N, K, TILE, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    printf("[DGEMM tiled] time=%.3f ms rate=%.2f GF/s diff=%.3e\n", ms,gflops,mad);
  }
  else if (task == 2) { // DGEMM cuBLAS
    CUDA_CHECK(hipEventRecord(e0, stream));
    cublas_rowmajor_dgemm(h, M, N, K, dA, dB, dC);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hC.data(), dC, hC.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hCref.data(), dCref, hCref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hC.data(), hCref.data(), hC.size());
    double gflops = (2.0*(double)M*N*K) / (ms*1e6);
    printf("[DGEMM cuBLAS] time=%.3f ms rate=%.2f GF/s diff=%.3e\n", ms,gflops,mad);
  }
  else if (task >= 3 && task <= 5) { // attention via DGEMM with your naive DGEMM or tiled DGEMM or cuBLAS
    int mode = (task==3)?0: (task==4)?1:2;
    CUDA_CHECK(hipEventRecord(e0, stream));
    attention_via_dgemm(dQ, dK, dV, dO, L, D, mode, h, stream);
    CUDA_CHECK(hipEventRecord(e1, stream)); CUDA_CHECK(hipEventSynchronize(e1));
    double ms = elapsed_ms(e0,e1);
    CUDA_CHECK(hipMemcpy(hO.data(), dO, hO.size()*sizeof(double), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(hOref.data(), dOref, hOref.size()*sizeof(double), hipMemcpyDeviceToHost));
    double mad = max_abs_diff_host(hO.data(), hOref.data(), hO.size());
    double flops = 4.0 * (double)L*L*D;
    double gflops = flops / (ms*1e6);
    const char* tag = (mode==0)?"naive":(mode==1)?"tiled":"cuBLAS";
    printf("[ATTN %s] time=%.3f ms rate=%.2f GF/s diff=%.3e\n", tag,ms,gflops,mad);
  }

  CUDA_CHECK(hipEventDestroy(e0)); CUDA_CHECK(hipEventDestroy(e1));
  CUDA_CHECK(hipFree(dA)); CUDA_CHECK(hipFree(dB)); CUDA_CHECK(hipFree(dC));
  CUDA_CHECK(hipFree(dCref));
  CUDA_CHECK(hipFree(dQ)); CUDA_CHECK(hipFree(dK)); CUDA_CHECK(hipFree(dV));
  CUDA_CHECK(hipFree(dO)); CUDA_CHECK(hipFree(dOref));
  CUBLAS_CHECK(hipblasDestroy(h)); CUDA_CHECK(hipStreamDestroy(stream));
  return 0;
}
