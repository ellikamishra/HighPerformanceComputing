#include "hip/hip_runtime.h"

#include "common.cuh"

// Each thread computes one C[row,col]
__global__ void dgemm_naive_kernel(const double* __restrict__ A,
                                   const double* __restrict__ B,
                                   double* __restrict__ C,
                                   int M, int N, int K)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= M || col >= N) return;
  double acc = 0.0;
  for (int t=0; t<K; ++t) {
    acc += A[(size_t)row*K + t] * B[(size_t)t*N + col];
  }
  C[(size_t)row*N + col] = acc;
}

// Host wrapper
void dgemm_naive_gpu(const double* dA, const double* dB, double* dC,
                     int M, int N, int K, hipStream_t stream)
{
  int bX = 16, bY = 16;
if (const char* s = std::getenv("NBX")) bX = std::max(1, atoi(s));
if (const char* s = std::getenv("NBY")) bY = std::max(1, atoi(s));
dim3 block(bX,bY);
  dim3 grid( (N + block.x - 1)/block.x, (M + block.y - 1)/block.y );
  dgemm_naive_kernel<<<grid, block, 0, stream>>>(dA, dB, dC, M, N, K);
  CUDA_CHECK(hipGetLastError());
}
